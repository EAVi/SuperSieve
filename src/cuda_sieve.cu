#include "hip/hip_runtime.h"
#include "cuda_sieve.h"
#include "cuda_sieve.cuh"

bool launch_cuda_sieve(char** binarray, const int & binarray_size, const int * primelist, const int & primelist_size, const int & start, const int & stepsize)
{
	hipError_t err;//error checking variable

	if(primelist == NULL || *binarray == NULL)
	{
		printf("NULL pointer passed\n");
		return 0;
	}
	if(primelist_size <= 0)
	{
		printf("primelist size is non-positive");
		return 0;
	}

	//memory size variables
	int bytesize = 1 + (binarray_size >> 4);
	size_t device_freemem = 0;
	size_t device_totalmem = 0;
	size_t membin = (sizeof(char) * bytesize);//output binarray size
	size_t memmeta = (sizeof(char) * (bytesize << 3));//matadata bigbinarr size
	size_t memlist = (sizeof(int) * primelist_size);//input primelist size
	size_t total_memneeded = membin + memlist;

	//the data to be allocated to device
	int * device_primelist = NULL;
	char * device_metaarray = NULL;
	char * device_binarray = NULL;

	//problem distribution across the device
	int blocksize = 512;//some arbitrary number that seems to be a sweetspot for the 1050Ti
	int numblocks = 1;//(primelist_size + blocksize - 1)/blocksize;//distribute across these many thread blocks

	//check to make sure there's enough memory on the device
	hipMemGetInfo(&device_freemem, &device_totalmem);
	if (total_memneeded > device_freemem)//not enough mem on device
	{
		printf("Not enough memory on CUDA device total: %i, free: %i, needed: %i\n",
			device_totalmem, device_freemem, total_memneeded);
		return false;
	}

	//allocate data onto to device
	err = hipMalloc(&device_primelist, memlist);
	if(err != hipSuccess)
	{
		printf("malloc failed on device primelist (%i)\n", err);
		return false;
	}
	err = hipMalloc(&device_binarray, membin);
	if(err != hipSuccess)
	{
		printf("malloc failed on device binarray (%i)\n", err);
		return false;
	}
	err = hipMalloc(&device_metaarray, memmeta);
	if(err != hipSuccess)
	{
		printf("malloc failed on device metaarray (%i)\n", err);
		return false;
	}

	//copy primelist to device
	err = hipMemcpy(device_primelist, primelist, memlist, hipMemcpyHostToDevice);
	if(err != hipSuccess)
	{
		printf("primelist memcpy failed(%i)\n", err);
		return false;
	}

	//call kernel
	cuda_sieve<<<numblocks,blocksize>>>(device_metaarray, binarray_size, device_primelist, primelist_size, start, stepsize);

	cuda_reduce<<<numblocks,blocksize>>>(device_metaarray, device_binarray, binarray_size);

	//copy binarray to cpu
	err = hipMemcpy(*binarray, device_binarray, membin, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		printf("binarray memcpy failed (%i)\n", err);
		return false;
	}

	//deallocate
	if (device_binarray != NULL)
	{
		err = hipFree(device_binarray);
		if (err != hipSuccess)
		{
			printf("device binarray free failed(%i)\n", err);
			return false;
		}
		device_binarray = NULL;
	}
	if (device_primelist != NULL)
	{
		err = hipFree(device_primelist);
		if (err != hipSuccess)
		{
			printf("device primelist free failed(%i)\n", err);
			return false;
		}
		device_primelist = NULL;
	}
	if (device_metaarray != NULL)
	{
		err = hipFree(device_metaarray);
		if (err != hipSuccess)
		{
			printf("device primelist free failed(%i)\n", err);
			return false;
		}
		device_metaarray = NULL;
	}
	return true;
}

__global__
void cuda_reduce(char* bigbinarray, char* smallbinarray, const int smallbinarray_size)
{
	int size =  smallbinarray_size >> 1; //cut the size in half to represent the bigbinarray size
	int bytestart = (threadIdx.x + blockDim.x * blockIdx.x) << 3;
	int bytestepsize = (blockDim.x * gridDim.x) << 3;
	//each thread writes in multiples of 8 to prevent race condition
	for(int i = bytestart; i < size; i += bytestepsize)
	{
//		printf("REDUCE tid %i, bid %i, i %i\n", threadIdx.x, blockIdx.x, i);
		for(int j = i, k = i + 8; j < k; ++j)
		{
			
			if (bigbinarray[j] == 0)//0 is prime, 1 is consonant; skip on prime
				continue;
			int oddnumber = 1 + (j << 1);//the odd number represented by the halved number
//			printf("reducing number %i\n", oddnumber);
			cuda_binarray_markconsonant(smallbinarray, smallbinarray_size, oddnumber);
		}
	}
}

__global__
void cuda_sieve(char* binarray, const int  binarray_size, const int * primelist, const int primelist_size, const int  start, const int  stepsize)
{
//	int metasize = binarray_size >> 1;
	int devicestepsize = gridDim.x * blockDim.x * stepsize;
	int devicestart = (threadIdx.x + start*blockDim.x) + blockDim.x * blockIdx.x * stepsize;
//printf("threadid %i start %i, step %i\n", threadIdx.x, devicestart, devicestepsize);
	//first prime should always be 2, otherwise, something's wrong
	if (primelist[0] != 2) 
	{
		printf("primelist was not properly passed\n");
		return;
	}
	int iterationcount = 0;
	for(int i = devicestart; i < primelist_size; i += devicestepsize)
	{
		iterationcount++;
		int a = primelist[i];
		if (a == 2) continue; //skip multiples of 2
		for(int j = a*3; j < binarray_size; j += a*2)
		{
			int metaindex = j >> 1;
			binarray[metaindex] = 1;
		}
	}
}

__device__
void cuda_binarray_markconsonant(char* binarray, const int & binarray_size, int i)
{
	//a cuda version of the markconsonant function in binarray.cpp
	if(i >= binarray_size)return;
	if(!(i & 1))return; //skip evens

	int shift = (i >> 1) & 7;
	binarray[i >> 4] |= (1 << shift);
}

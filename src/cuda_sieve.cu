#include "hip/hip_runtime.h"
#include "cuda_sieve.h"
#include "cuda_sieve.cuh"

bool launch_cuda_sieve(char** binarray, const int & binarray_size, const int * primelist, const int & primelist_size, const int & start, const int & stepsize)
{
	hipError_t err;//error checking variable

	if(primelist == NULL || *binarray == NULL)
	{
		printf("NULL pointer passed\n");
		return 0;
	}
	if(primelist_size <= 0)
	{
		printf("primelist size is non-positive");
		return 0;
	}

	//memory size variables
	int bytesize = 1 + (binarray_size >> 4);
	size_t device_freemem = 0;
	size_t device_totalmem = 0;
	size_t membin = (sizeof(char) * bytesize);
	size_t memlist = (sizeof(int) * primelist_size);
	size_t total_memneeded = membin + memlist;

	//the data to be allocated to device
	int * device_primelist = NULL;
	char * device_binarray = NULL;

	//check to make sure there's enough memory on the device
	hipMemGetInfo(&device_freemem, &device_totalmem);
	if (total_memneeded > device_freemem)//not enough mem on device
	{
		printf("Not enough memory on CUDA device total: %i, free: %i, needed: %i\n",
			device_totalmem, device_freemem, total_memneeded);
		return false;
	}

	//allocate data onto to device
	err = hipMalloc(&device_primelist, memlist);
	if(err != hipSuccess)
	{
		printf("malloc failed on device primelist (%i)\n", err);
		return false;
	}
	err = hipMalloc(&device_binarray, membin);
	if(err != hipSuccess)
	{
		printf("malloc failed on device binarray (%i)\n", err);
		return false;
	}

	//copy primelist to device
	err = hipMemcpy(device_primelist, primelist, memlist, hipMemcpyHostToDevice);
	if(err != hipSuccess)
	{
		printf("primelist memcpy failed(%i)\n", err);
		return false;
	}

	//call kernel
	cuda_sieve<<<1,1>>>(device_binarray, binarray_size, device_primelist, primelist_size, start, stepsize);
	hipDeviceSynchronize();

	//copy binarray to cpu
	err = hipMemcpy(*binarray, device_binarray, membin, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		printf("binarray memcpy failed (%i)\n", err);
		return false;
	}

	//deallocate
	if (device_binarray != NULL)
	{
		err = hipFree(device_binarray);
		if (err != hipSuccess)
		{
			printf("device binarray free failed(%i)\n", err);
			return false;
		}
		device_binarray = NULL;
	}
	if (device_primelist != NULL)
	{
		err = hipFree(device_primelist);
		if (err != hipSuccess)
		{
			printf("device primelist free failed(%i)\n", err);
			return false;
		}
		device_primelist = NULL;
	}
	return true;
}

__global__
void cuda_sieve(char* binarray, const int  binarray_size, const int * primelist, const int primelist_size, const int  start, const int  stepsize)
{
	//first prime should always be 2, otherwise, something's wrong
	if (primelist[0] != 2) 
	{
		printf("primelist was not properly passed\n");
		return;
	}

	for(int i = start; i < primelist_size; i += stepsize)
	{
		int a = primelist[i];
		if (a == 2) continue; //skip evens
		for(int j = a*3; j < binarray_size; j += a*2)
		{
			cuda_binarray_markconsonant(binarray, binarray_size, j);
		}
	}
}

__device__
void cuda_binarray_markconsonant(char* binarray, const int & binarray_size, int i)
{
	//a cuda version of the markconsonant function in binarray.cpp
	if(i >= binarray_size)return;
	if(!(i & 1))return; //skip evens

	int shift = (i >> 1) & 7;
	binarray[i >> 4] |= (1 << shift);
}
